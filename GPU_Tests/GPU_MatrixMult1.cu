#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include "hip/hip_runtime.h"
#include ""

__global__ void matrixMultiplyKernel(int* A, int* B, int* C, int width) {
    int i = threadIdx.y;
    int j = threadIdx.x;
    int value = 0;

    for (int k = 0; k < width; k++) {
        value += A[i * width + k] * B[k * width + j];
    }
    C[i * width + j] = value;
}

void matrixMultiply(const int* A, const int* B, int* C, int width) {
    int* dev_A = 0;
    int* dev_B = 0;
    int* dev_C = 0;

    hipMalloc((void**)&dev_A, width * width * sizeof(int));
    hipMalloc((void**)&dev_B, width * width * sizeof(int));
    hipMalloc((void**)&dev_C, width * width * sizeof(int));

    hipMemcpy(dev_A, A, width * width * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_B, B, width * width * sizeof(int), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(width, width);
    matrixMultiplyKernel << <1, threadsPerBlock >> > (dev_A, dev_B, dev_C, width);

    hipMemcpy(C, dev_C, width * width * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);
}

int main() {
    const int width = 2;
    int A[2][2] = { {1, 2}, {3, 4} };
    int B[2][2] = { {2, 0}, {1, 2} };
    int C[2][2] = { 0 };

    auto start = std::chrono::high_resolution_clock::now();
    matrixMultiply(&A[0][0], &B[0][0], &C[0][0], width);
    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);

    std::cout << "Result Matrix:\n";
    for (int i = 0; i < 2; i++) {
        for (int j = 0; j < 2; j++) {
            std::cout << C[i][j] << " ";
        }
        std::cout << "\n";
    }
    std::cout << "Execution time: " << duration.count() << " microseconds\n";

    return 0;
}
